#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "functions.cuh"
#include <math.h>

extern int matDim;

int main(int argc, char ** argv){
	
	/*----------INIT---------------------*/
	char *mode;
	mode = (char *) malloc(strlen(argv[1])+1);
	strcpy(mode, argv[1]);
	char cpu[4] = "cpu";
	char gpu[4] = "gpu";

	int n = atoi(argv[2]);
	int matDim = pow(2,n) + 1;
	int nbMatEl = matDim * matDim;;
	unsigned char *data, *devData, *tabDepth, *devTabDepth;
	int4 *_tabParents, *dev_tabParents;
	
	/*Init data*/
	data = (unsigned char *) calloc(matDim * matDim, sizeof *data);
	data[0] = data[matDim - 1] = data[ (matDim - 1) * matDim] = data[matDim*matDim - 1] = 1;
    data[matDim / 2 * matDim + matDim / 2] = matDim / 2 + 1;

	/*-------------CPU MODE--------------------*/

	if(strcmp(mode,cpu) == 0){
		initTime();
		diamontCPU(matDim, matDim, data);
		printf("%d\t time cpu : %F\n", nbMatEl, getTime());
	}

	/*-------------GPU MODE--------------------*/

	if(strcmp(mode,gpu) == 0){

		/*mat profondeur*/
		tabDepth = (unsigned char *) calloc(matDim * matDim, sizeof *tabDepth);
		_tabParents = (int4 *) calloc(matDim * matDim, sizeof *_tabParents);
		tabDepth[0] = tabDepth[matDim - 1] = tabDepth[(matDim - 1) * matDim] = tabDepth[matDim * matDim-1] = 0;

		/*Remplissage mat de parents et profondeur*/
		initArrays(tabDepth, _tabParents, matDim, matDim);

		/*Affichage profondeur*/
		// char matProfName[] = "profondeur";
		// displayMat(tabDepth, matProfName, matDim);	

		hipMalloc(&devData, nbMatEl* sizeof *devData);
		hipMalloc(&devTabDepth, nbMatEl* sizeof *devTabDepth);
		hipMalloc(&dev_tabParents, nbMatEl* sizeof *dev_tabParents);

		hipMemcpy(devData, data, nbMatEl* sizeof *data, hipMemcpyHostToDevice);
		hipMemcpy(devTabDepth, tabDepth, nbMatEl* sizeof *tabDepth, hipMemcpyHostToDevice);
		hipMemcpy(dev_tabParents, _tabParents, nbMatEl* sizeof *_tabParents, hipMemcpyHostToDevice);
		
		int _maxProf = maxProf(tabDepth, matDim);
		printf("Max %d\n", _maxProf);

		int th = _maxProf;
		dim3 dimBlock(th, th, 1);
		dim3 dimGrid((matDim / dimBlock.x)+1, (matDim / dimBlock.y)+1, 1);
		
		initTime();
		for(int i = 0; i<=_maxProf; ++i){
			diamont<<<dimGrid, dimBlock, 0>>>(devData, devTabDepth, dev_tabParents, i, matDim);
		}
		printf("%d\t time gpu : %F\n", nbMatEl, getTime());

		hipMemcpy(data, devData, nbMatEl* sizeof *data, hipMemcpyDeviceToHost);
	}

	// char matDatasName[] = "datas";
    // displayMat(data, matDatasName, matDim);
	
	return 0;
}