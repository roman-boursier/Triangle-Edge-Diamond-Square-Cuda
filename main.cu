#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "functions.cuh"
#include <math.h>
#include <opencv2/opencv.hpp>

using namespace cv;

int main(int argc, char ** argv){
	
	/*----------INIT---------------------*/

	//Mode arg
	char *mode;
	mode = (char *) malloc(strlen(argv[1])+1);
	strcpy(mode, argv[1]);
	char cpu[4] = "cpu";
	char gpu[4] = "gpu";

	//N arg
	int n = atoi(argv[2]);

	//Filename arg
	char *filename;
	filename = (char *) malloc(strlen(argv[3])+1);
	strcpy(filename, argv[3]);

	int matDim = pow(2,n) + 1;
	int nbMatEl = matDim * matDim;
	unsigned char *data, *devData, *tabDepth, *devTabDepth;
	int4 *_tabParents, *dev_tabParents;
	
	//Img output
	Mat img(matDim, matDim, CV_8UC1);
	const int grayBytes = img.step * img.rows;
	unsigned char *d_input, *d_img;

	/*Init data*/
	data = (unsigned char *) calloc(matDim * matDim, sizeof *data);
	data[0] = data[matDim - 1] = data[ (matDim - 1) * matDim] = data[matDim*matDim - 1] = 1;
    data[matDim / 2 * matDim + matDim / 2] = matDim / 2 + 1;

	/*-------------CPU MODE--------------------*/

	if(strcmp(mode,cpu) == 0){
		initTime();
		diamontCPU(matDim, matDim, data);
		printf("%d\t%F\n", n, getTime());
	}

	/*-------------GPU MODE--------------------*/

	if(strcmp(mode,gpu) == 0){

		/*mat profondeur*/
		tabDepth = (unsigned char *) calloc(matDim * matDim, sizeof *tabDepth);
		_tabParents = (int4 *) calloc(matDim * matDim, sizeof *_tabParents);
		tabDepth[0] = tabDepth[matDim - 1] = tabDepth[(matDim - 1) * matDim] = tabDepth[matDim * matDim-1] = 0;

		/*Remplissage mat de parents et profondeur*/
		initArrays(tabDepth, _tabParents, matDim, matDim);

		/*Affichage profondeur*/
		// char matProfName[] = "profondeur";
		// displayMat(tabDepth, matProfName, matDim);	

		hipMalloc(&devData, nbMatEl* sizeof *devData);
		hipMalloc(&devTabDepth, nbMatEl* sizeof *devTabDepth);
		hipMalloc(&dev_tabParents, nbMatEl* sizeof *dev_tabParents);
		hipMalloc(&d_img, grayBytes);

		hipMemcpy(devData, data, nbMatEl* sizeof *data, hipMemcpyHostToDevice);
		hipMemcpy(devTabDepth, tabDepth, nbMatEl* sizeof *tabDepth, hipMemcpyHostToDevice);
		hipMemcpy(dev_tabParents, _tabParents, nbMatEl* sizeof *_tabParents, hipMemcpyHostToDevice);
		hipMemcpy(d_input, img.ptr(), grayBytes, hipMemcpyHostToDevice);
		
		int _maxProf = maxProf(tabDepth, matDim);
		// printf("Max %d\n", _maxProf);

		int th = _maxProf;
		dim3 dimBlock(th, th, 1);
		dim3 dimGrid((matDim / dimBlock.x)+1, (matDim / dimBlock.y)+1, 1);
		
		initTime();
		for(int i = 0; i<=_maxProf; ++i){
			diamont<<<dimGrid, dimBlock, 0>>>(devData, devTabDepth, dev_tabParents, i, matDim);
			//diamontImg<<<dimGrid, dimBlock, 0>>>(devData, d_img, devTabDepth, dev_tabParents, i, matDim);
		}
		printf("%d\t%F\n", n, getTime());
		//printf("%d\t time gpu : %F\n", nbMatEl, getTime());

		hipMemcpy(data, devData, nbMatEl* sizeof *data, hipMemcpyDeviceToHost);
		hipMemcpy(img.ptr(), d_img, grayBytes, hipMemcpyDeviceToHost);

		//cv::imwrite(filename, img);
	}

	// char matDatasName[] = "datas";
	// displayMat(data, matDatasName, matDim);
	
	return 0;
}