#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "functions.cuh"
#include <math.h>
#include <opencv2/opencv.hpp>

using namespace cv;

int main(int argc, char ** argv){
	
	/*----------INIT---------------------*/

	//Mode arg
	char *mode;
	mode = (char *) malloc(strlen(argv[1])+1);
	strcpy(mode, argv[1]);
	char cpu[4] = "cpu";
	char gpu[4] = "gpu";

	//N arg
	int n = atoi(argv[2]);

	//Filename arg
	char *filename;
	filename = (char *) malloc(strlen(argv[3])+1);
	strcpy(filename, argv[3]);

	int matDim = pow(2,n) + 1;
	int nbMatEl = matDim * matDim;
	unsigned char *data, *devData, *tabDepth, *devTabDepth;
	int4 *_tabParents, *dev_tabParents;
	
	/*Init data*/
	data = (unsigned char *) calloc(matDim * matDim, sizeof *data);
	data[0] = data[matDim - 1] = data[ (matDim - 1) * matDim] = data[matDim*matDim - 1] = 1;
    data[matDim / 2 * matDim + matDim / 2] = 255;

	/*-------------CPU MODE--------------------*/

	if(strcmp(mode,cpu) == 0){
		//initTime();

		diamontCPU(matDim, matDim, data);

		//Img
		Mat img(matDim, matDim, CV_8UC1, Scalar(0));
		for(int y = 0; y < matDim; y ++){
			for(int x = 0; x < matDim; x ++){
				img.at<uchar>(y,x) = data[y * matDim + x];
			}
		}
		//makprintf("%d\t%F\n", n, getTime());

		cv::imwrite(filename, img);
	}

	/*-------------GPU MODE--------------------*/

	if(strcmp(mode,gpu) == 0){

		/*mat profondeur*/
		tabDepth = (unsigned char *) calloc(matDim * matDim, sizeof *tabDepth);
		_tabParents = (int4 *) calloc(matDim * matDim, sizeof *_tabParents);
		tabDepth[0] = tabDepth[matDim - 1] = tabDepth[(matDim - 1) * matDim] = tabDepth[matDim * matDim-1] = 0;

		_tabParents[0].x = _tabParents[matDim - 1].x = _tabParents[(matDim - 1) * matDim].x = _tabParents[matDim * matDim-1].x = -1;
		_tabParents[0].y = _tabParents[matDim - 1].y = _tabParents[(matDim - 1) * matDim].y = _tabParents[matDim * matDim-1].y = -1;
		_tabParents[0].z = _tabParents[matDim - 1].z = _tabParents[(matDim - 1) * matDim].z = _tabParents[matDim * matDim-1].z = -1;
		_tabParents[0].w = _tabParents[matDim - 1].w = _tabParents[(matDim - 1) * matDim].w = _tabParents[matDim * matDim-1].w = -1;
		//tabDepth[matDim / 2 * matDim + matDim / 2] = 9;
		/*Remplissage mat de parents et profondeur*/
		initArrays(tabDepth, _tabParents, matDim, matDim);

		/*Affichage profondeur*/
		char matProfName[] = "profondeur";
		displayMat(tabDepth, matProfName, matDim);	
		//Img output

		Mat img(matDim, matDim, CV_8UC1);
		const int grayBytes = img.step * img.rows;
		unsigned char *d_input, *d_img;
		hipMalloc(&d_img, nbMatEl* sizeof grayBytes);
		hipMemcpy(d_input, img.ptr(), grayBytes, hipMemcpyHostToDevice);

		hipMalloc(&devData, nbMatEl* sizeof *devData);
		hipMalloc(&devTabDepth, nbMatEl* sizeof *devTabDepth);
		hipMalloc(&dev_tabParents, nbMatEl* sizeof *dev_tabParents);
		
		hipMemcpy(devData, data, nbMatEl* sizeof *data, hipMemcpyHostToDevice);
		hipMemcpy(devTabDepth, tabDepth, nbMatEl* sizeof *tabDepth, hipMemcpyHostToDevice);
		hipMemcpy(dev_tabParents, _tabParents, nbMatEl* sizeof *_tabParents, hipMemcpyHostToDevice);
		
		
		int _maxProf = maxProf(tabDepth, matDim);

		int th = _maxProf;
		printf("%d", _maxProf);
		dim3 dimBlock(th, th, 1);
		dim3 dimGrid((matDim / dimBlock.x)+1, (matDim / dimBlock.y)+1, 1);
		
		//initTime();
		for(int i = 1; i<=_maxProf; i++){
			diamontImg<<<dimGrid, dimBlock, 0>>>(devData, d_img, devTabDepth, dev_tabParents, i, matDim);
		}

		hipMemcpy(data, devData, nbMatEl* sizeof *data, hipMemcpyDeviceToHost);
		hipMemcpy(img.ptr(), d_img, grayBytes, hipMemcpyDeviceToHost);

		cv::imwrite(filename, img);
	}

	// char matDatasName[] = "datas";
	// displayMat(data, matDatasName, matDim);

	hipFree(devData); free(data);
	hipFree(devTabDepth); free(tabDepth);
	
	return 0;
}