
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

static struct timeval ti;

bool IN(int x, int y, int w, int h) {
	return (x) >= 0 && (y) >= 0 && (x) < (w) && (y) < (h);
}

int maxProf(unsigned char * arr, int matDim){
	int max_value = 0;

	for (int i = 0; i < matDim; ++i) {
		if (arr[i] > max_value)
			max_value = arr[i];
	}

	return max_value;
}

void displayMat(unsigned char * arr, char * name, int matDim){
	printf("Matrice de %s \n \n", name);
	for(int y = 0; y < matDim; ++y){
    	for(int x = 0; x < matDim; ++x){
        	printf("%d ", (int)arr[y * matDim + x]);
      	}
		printf("\n");
	}
	printf("\n");	
}

void initArraysDs(unsigned char * _tabDepth, int4 * _tabParents, int w, int h) {
	int sw1 = w - 1, sw2 = sw1 >> 1;
	int sh1 = h - 1, sh2 = sh1 >> 1;
	int niveau = 1;
	while(sw2){
		for(int y = sh2; y < h; y += sh1){
			for(int x = sw2; x < w; x += sw1){
				if(_tabDepth[y * w + x]) continue;
				_tabDepth[y * w + x] = niveau;
				_tabParents[y * w + x].x = (y - sh2) * w + x - sw2;
 				_tabParents[y * w + x].y = (y - sh2) * w + x + sw2;
 				_tabParents[y * w + x].z = (y + sh2) * w + x + sw2;
				_tabParents[y * w + x].w = (y + sh2) * w + x - sw2;
 			}

		}
		niveau++; 
		for(int y = 0; y < h; y += sh1){
			for(int x = sw2; x < w; x += sw1){
				int yp, xp;
				if(_tabDepth[y * w + x]) continue;
				yp = y - sh2; xp = x; // parent haut
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].x = yp * w + xp;
				} else{
					_tabParents[y * w + x].x = -1;
				}
				yp = y; xp = x + sw2; // parent droit
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].y = yp * w + xp;
				} else{
					_tabParents[y * w + x].y = -1;
				}
				yp = y + sh2; xp = x; // parent bas
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].z = yp * w + xp;
				} else{
					_tabParents[y * w + x].z = -1;
				}
				yp = y; xp = x - sw2; // parent gauche
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].w = yp * w + xp;
				} else{
					_tabParents[y * w + x].w = -1;
				}
				_tabDepth[y * w + x] = niveau;
 			}
			
		}
		for(int y = sh2; y < h; y += sh1){
			for(int x = 0; x < w; x += sw1){
				int yp, xp;
				if(_tabDepth[y * w + x]) continue;
				yp = y - sh2; xp = x; // parent haut
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].x = yp * w + xp;
				} else{
					_tabParents[y * w + x].x = -1;
				}
				yp = y; xp = x + sw2; // parent droit
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].y = yp * w + xp;
				} else{
					_tabParents[y * w + x].y = -1;
				}
				yp = y + sh2; xp = x; // parent bas
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].z = yp * w + xp;
				} else{
					_tabParents[y * w + x].z = -1;
				}
				yp = y; xp = x - sw2; // parent gauche
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].w = yp * w + xp;
				} else{
					_tabParents[y * w + x].w = -1;
				}
				_tabDepth[y * w + x] = niveau;
 			}
			
		}
		niveau ++;
		sw1 = sw2;
		sw2 >>= 1;
		sh1 = sh2;
		sh2 >>= 1;
	}
}

void initArraysTe(unsigned char * _tabDepth, int4 * _tabParents, int x0, int y0, int stride, int sx, int sy, int p) {
	int w_2 = sx >> 1, h_2 = sy >> 1, j;
    int x[9] = { x0, x0 + sx, x0 + sx,
    x0, x0 + w_2, x0 + sx, x0 + w_2,
    x0, x0 + w_2 };
    int y[9] = { y0,
    y0, y0 + sy, y0 + sy,
    y0, y0 + h_2, y0 + sy, y0 + h_2, y0 + h_2 };

    for(int i = 0, i1 = 1; i < 4; ++i, i1 = (i1 + 1) % 4) {
		if(_tabDepth[j = y[i + 4] * stride + x[i + 4]]) continue;
			//Parents			
			_tabParents[j].x = y[i] * stride + x[i];
			_tabParents[j].y = y[i1] * stride + x[i1];
			_tabParents[j].z = -1;
			_tabParents[j].w = -1;
			//prof
			_tabDepth[j] = p;
        }

		if(!_tabDepth[j = y[8] * stride + x[8]]) {
            for(int i = 0; i < 4; ++i) {
				//prof
				_tabDepth[j] = p;
				//Tab parent
				if(i == 1){
					_tabParents[j].x = y[i] * stride + x[i];
				} else if(i == 2){
					_tabParents[j].y = y[i] * stride + x[i];
				} else if(i == 3) {
					_tabParents[j].z = y[i] * stride + x[i];
				} else {
					_tabParents[j].w = y[i] * stride + x[i];
				}
			}
			
        }
        if(w_2 > 1) {
            initArraysTe(_tabDepth, _tabParents, x[0], y[0], stride, w_2, h_2, ++p);
            initArraysTe(_tabDepth, _tabParents, x[4], y[4], stride, w_2, h_2, p);
            initArraysTe(_tabDepth, _tabParents, x[8], y[8], stride, w_2, h_2, p);
            initArraysTe(_tabDepth, _tabParents, x[7], y[7], stride, w_2, h_2, p);
    }
}

void triangleEdgeCPU(unsigned char * data, int x0, int y0, int stride, int sx, int sy, int p) {
	int niveau = 1;
	
    int w_2 = sx >> 1, h_2 = sy >> 1, j;
    int x[9] = { x0, x0 + sx, x0 + sx,
    x0, x0 + w_2, x0 + sx, x0 + w_2,
    x0, x0 + w_2 };
    int y[9] = { y0,
    y0, y0 + sy, y0 + sy,
    y0, y0 + h_2, y0 + sy, y0 + h_2, y0 + h_2 };

    for(int i = 0, i1 = 1; i < 4; ++i, i1 = (i1 + 1) % 4) {
        if(data[j = y[i + 4] * stride + x[i + 4]]) continue;
            data[j] = ((int)data[y[i] * stride + x[i]] +
            data[y[i1] * stride + x[i1]]) >> 1;
        }
        if(!data[j = y[8] * stride + x[8]]) {
            int v = 0;
            for(int i = 0; i < 4; ++i)
            v += data[y[i] * stride + x[i]];
            data[j] = v >> 2;
        }
        if(w_2 > 1) {
            triangleEdgeCPU(data, x[0], y[0], stride, w_2, h_2, ++p);
            triangleEdgeCPU(data, x[4], y[4], stride, w_2, h_2, p);
            triangleEdgeCPU(data, x[8], y[8], stride, w_2, h_2, p);
            triangleEdgeCPU(data, x[7], y[7], stride, w_2, h_2, p);
    }
}

void diamontCPU(int w, int h, unsigned char * data) {
	int sw1 = w - 1, sw2 = sw1 >> 1;
	int sh1 = h - 1, sh2 = sh1 >> 1;

	while(sw2){
		for(int y = sh2; y < h; y += sh1){
			for(int x = sw2; x < w; x += sw1){
				if(data[y * w + x]) continue;

				data[y * w + x] = (data[(y - sh2) * w + x - sw2] +
				data[(y - sh2) * w + x + sw2] +
				data[(y + sh2) * w + x + sw2] +
				data[(y + sh2) * w + x - sw2]) >> 2;
 			}
		}

		for(int y = 0; y < h; y += sh1){
			for(int x = sw2; x < w; x += sw1){
				int yp, xp;
				int nbp = 0, v = 0;

				if(data[y * w + x]) continue;
				yp = y - sh2; xp = x; // parent haut
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}

				yp = y; xp = x + sw2; // parent droit
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}
				yp = y + sh2; xp = x; // parent bas
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}
				yp = y; xp = x - sw2; // parent gauche
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				} 
				data[y * w + x] = v / nbp;
 			}	
		}
		for(int y = sh2; y < h; y += sh1){
			for(int x = 0; x < w; x += sw1){
				int yp, xp;
				int nbp = 0, v = 0;

				if(data[y * w + x]) continue;
				yp = y - sh2; xp = x; // parent haut
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}
				yp = y; xp = x + sw2; // parent droit
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}
				yp = y + sh2; xp = x; // parent bas
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}

				yp = y; xp = x - sw2; // parent gauche
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}
				data[y * w + x] = v / nbp;
 			}
			
		}
		sw1 = sw2;
		sw2 >>= 1;
		sh1 = sh2;
		sh2 >>= 1;
	}
}

__global__ void generateImg(unsigned char * data, unsigned char * img, unsigned char * tabDepth, int4 * _tabParents, int i, int tailleTab) {
	int thx = blockIdx.x * blockDim.x + threadIdx.x;
	int thy = blockIdx.y * blockDim.y + threadIdx.y;
	int ThId = thy * tailleTab + thx;
	int nbPar = 0;

	if(data[ThId] == 0 && tabDepth[ThId] == i  && i != 1) {
	
		if(_tabParents[ThId].x != -1) nbPar ++;
		if(_tabParents[ThId].y != -1) nbPar ++;
		if(_tabParents[ThId].z != -1) nbPar ++;
		if(_tabParents[ThId].w != -1) nbPar ++;
		
		data[ThId] = (data[_tabParents[ThId].x] + data[_tabParents[ThId].y] + data[_tabParents[ThId].z] + data[_tabParents[ThId].w]) / nbPar;

		img[ThId] = data[ThId];
	}
	
}

void initTime(void) {
    gettimeofday(&ti, (struct timezone*) 0);
}

double getTime(void) {
    struct timeval t;
    double diff;
    gettimeofday(&t, (struct timezone*) 0);
    diff = (t.tv_sec - ti.tv_sec) * 1000000
    + (t.tv_usec - ti.tv_usec);
    return diff/1000.;
}