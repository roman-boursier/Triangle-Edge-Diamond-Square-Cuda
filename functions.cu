
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

bool IN(int x, int y, int w, int h) {
	return (x) >= 0 && (y) >= 0 && (x) < (w) && (y) < (h);
}

int maxProf(unsigned char * arr, int matDim){
	int max_value = 0;

	for (int i = 0; i < matDim; ++i) {
		if (arr[i] > max_value)
			max_value = arr[i];
	}

	return max_value;
}

void displayMat(unsigned char * arr, char * name, int matDim){
	printf("Matrice de %s \n \n", name);
	for(int y = 0; y < matDim; ++y){
    	for(int x = 0; x < matDim; ++x){
        	printf("%d ", (int)arr[y * matDim + x]);
      	}
		printf("\n");
	}
	printf("\n");	
}

void initArrays(unsigned char * _tabDepth, int4 * _tabParents, int w, int h) {
	int sw1 = w - 1, sw2 = sw1 >> 1;
	int sh1 = h - 1, sh2 = sh1 >> 1;
	int niveau = 1;
	while(sw2){
		for(int y = sh2; y < h; y += sh1){
			for(int x = sw2; x < w; x += sw1){
				if(_tabDepth[y * w + x]) continue;
				_tabDepth[y * w + x] = niveau;
				_tabParents[y * w + x].x = (y - sh2) * w + x - sw2;
 				_tabParents[y * w + x].y = (y - sh2) * w + x + sw2;
 				_tabParents[y * w + x].z = (y + sh2) * w + x + sw2;
				_tabParents[y * w + x].w = (y + sh2) * w + x - sw2;
 			}

		}
		niveau++; 
		for(int y = 0; y < h; y += sh1){
			for(int x = sw2; x < w; x += sw1){
				int yp, xp;
				if(_tabDepth[y * w + x]) continue;
				yp = y - sh2; xp = x; // parent haut
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].x = yp * w + xp;
				} else{
					_tabParents[y * w + x].x = -1;
				}
				yp = y; xp = x + sw2; // parent droit
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].y = yp * w + xp;
				} else{
					_tabParents[y * w + x].y = -1;
				}
				yp = y + sh2; xp = x; // parent bas
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].z = yp * w + xp;
				} else{
					_tabParents[y * w + x].z = -1;
				}
				yp = y; xp = x - sw2; // parent gauche
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].w = yp * w + xp;
				} else{
					_tabParents[y * w + x].w = -1;
				}
				_tabDepth[y * w + x] = niveau;
 			}
			
		}
		for(int y = sh2; y < h; y += sh1){
			for(int x = 0; x < w; x += sw1){
				int yp, xp;
				if(_tabDepth[y * w + x]) continue;
				yp = y - sh2; xp = x; // parent haut
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].x = yp * w + xp;
				} else{
					_tabParents[y * w + x].x = -1;
				}
				yp = y; xp = x + sw2; // parent droit
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].y = yp * w + xp;
				} else{
					_tabParents[y * w + x].y = -1;
				}
				yp = y + sh2; xp = x; // parent bas
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].z = yp * w + xp;
				} else{
					_tabParents[y * w + x].z = -1;
				}
				yp = y; xp = x - sw2; // parent gauche
				if(IN(xp, yp, w, h)){
					_tabParents[y * w + x].w = yp * w + xp;
				} else{
					_tabParents[y * w + x].w = -1;
				}
				_tabDepth[y * w + x] = niveau;
 			}
			
		}
		niveau ++;
		sw1 = sw2;
		sw2 >>= 1;
		sh1 = sh2;
		sh2 >>= 1;
	}
}

void diamontCPU(int w, int h, unsigned char * data) {
	int sw1 = w - 1, sw2 = sw1 >> 1;
	int sh1 = h - 1, sh2 = sh1 >> 1;

	while(sw2){
		for(int y = sh2; y < h; y += sh1){
			for(int x = sw2; x < w; x += sw1){
				if(data[y * w + x]) continue;

				data[y * w + x] = (data[(y - sh2) * w + x - sw2] +
				data[(y - sh2) * w + x + sw2] +
				data[(y + sh2) * w + x + sw2] +
				data[(y + sh2) * w + x - sw2]) >> 2;
 			}
		}

		for(int y = 0; y < h; y += sh1){
			for(int x = sw2; x < w; x += sw1){
				int yp, xp;
				int nbp = 0, v = 0;

				if(data[y * w + x]) continue;
				yp = y - sh2; xp = x; // parent haut
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}

				yp = y; xp = x + sw2; // parent droit
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}
				yp = y + sh2; xp = x; // parent bas
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}
				yp = y; xp = x - sw2; // parent gauche
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				} 
				data[y * w + x] = v / nbp;
 			}	
		}
		for(int y = sh2; y < h; y += sh1){
			for(int x = 0; x < w; x += sw1){
				int yp, xp;
				int nbp = 0, v = 0;

				if(data[y * w + x]) continue;
				yp = y - sh2; xp = x; // parent haut
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}
				yp = y; xp = x + sw2; // parent droit
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}
				yp = y + sh2; xp = x; // parent bas
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}

				yp = y; xp = x - sw2; // parent gauche
				if(IN(xp, yp, w, h)){
					++nbp;
					v += data[yp * w + xp] ;
				}
				data[y * w + x] = v / nbp;
 			}
			
		}
		sw1 = sw2;
		sw2 >>= 1;
		sh1 = sh2;
		sh2 >>= 1;
	}
}

__global__ void diamont(unsigned char * data, unsigned char * tabDepth, int4 * _tabParents, int i, int tailleTab) {
	int thx = blockIdx.x * blockDim.x + threadIdx.x;
	int thy = blockIdx.y * blockDim.y + threadIdx.y;
	int ThId = thy * tailleTab + thx;
	int nbPar = 0;

	if(tabDepth[ThId] == i  && i != 1) {

		if(_tabParents[ThId].x != -1) nbPar ++;
		if(_tabParents[ThId].y != -1) nbPar ++;
		if(_tabParents[ThId].z != -1) nbPar ++;
		if(_tabParents[ThId].w != -1) nbPar ++;
		
		data[ThId] = (data[_tabParents[ThId].x] + data[_tabParents[ThId].y] + data[_tabParents[ThId].z] + data[_tabParents[ThId].w]);
		
		(nbPar == 0) ? data[ThId] = 1 : data[ThId] /= nbPar;
	}
}